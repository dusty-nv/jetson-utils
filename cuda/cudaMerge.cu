#include "hip/hip_runtime.h"
#include "cudaMerge.h"



// gpuMerge.
// * Merge an image on the GPU (supports RGB/BGR, RGBA/BGRA from some single color planes(using GRAY format))
template<typename T, int CH>
__global__ void gpuMerge(T *input0, T *input1, T *input2, T *input3, T *output, size_t width, size_t height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
		return;

	const T p0 = input0[y * width + x];
	const T p1 = input1[y * width + x];
	const T p2 = input2[y * width + x];
	const T p3 = (CH == 4) ? input3[y * width + x] : T(0);

	output[(y * width + x) * CH + 0] = p0;
	output[(y * width + x) * CH + 1] = p1;
	output[(y * width + x) * CH + 2] = p2;
	if (CH == 4) output[(y * width + x) * CH + 3] = p3;
}

// gpuMerge.
// * Merge an image on the GPU (supports RGB/BGR, RGBA/BGRA from 3 colors and alpha plane)
template<typename T, typename S, typename R>
__global__ void gpuMerge(T *input_color, S *input_alpha, R *output, size_t width, size_t height)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
		return;

	T pix_color = input_color[y * width + x];
	S pix_alpha = input_alpha[y * width + x];
	R pix = make_vec<R>(pix_color.x, pix_color.y, pix_color.z, pix_alpha);

	output[y * width + x] = pix;
}

// launchMerge
// * Merge an image on the GPU (supports RGB/BGR, RGBA/BGRA from some single color planes(using GRAY format))
template<typename T, int CH>
static hipError_t launchMerge(T **input, T *output, size_t width, size_t height)
{
	if( !input[0] || !input[1] || !input[2] || (CH == 4 ? !input[3] : false) || !output )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(32, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	gpuMerge<T, CH><<<gridDim, blockDim>>>(input[0], input[1], input[2], (CH == 4) ? input[3] : nullptr, output, width, height);

	return CUDA(hipGetLastError());
}

// launchMerge
// * Merge an image on the GPU (supports RGB/BGR, RGBA/BGRA from 3 colors and alpha plane)
template<typename T, typename S, typename R>
static hipError_t launchMerge(T *input_color, S *input_alpha, R *output, size_t width, size_t height)
{
	if( !input_color || !input_alpha || !output )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(32, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	gpuMerge<T, S, R><<<gridDim, blockDim>>>(input_color, input_alpha, output, width, height);

	return CUDA(hipGetLastError());
}

//-----------------------------------------------------------------------------------
hipError_t cudaMerge(void **input, void *output, size_t width, size_t height, imageFormat format)
{
	if( format == IMAGE_RGB8 || format == IMAGE_BGR8 )
		return launchMerge<uchar, 3>((uchar **)input, (uchar *)output, width, height);
	else if( format == IMAGE_RGBA8 || format == IMAGE_BGRA8 )
		return launchMerge<uchar, 4>((uchar **)input, (uchar *)output, width, height);
	else if( format == IMAGE_RGB32F || format == IMAGE_BGR32F )
		return launchMerge<float, 3>((float **)input, (float *)output, width, height);
	else if( format == IMAGE_RGBA32F || format == IMAGE_BGRA32F )
		return launchMerge<float, 4>((float **)input, (float *)output, width, height);

	LogError(LOG_CUDA "cudaMerge() -- invalid image format '%s'\n", imageFormatToStr(format));
	LogError(LOG_CUDA "                supported formats are:\n");
	LogError(LOG_CUDA "                    * rgb8, bgr8\n");
	LogError(LOG_CUDA "                    * rgba8, bgra8\n");
	LogError(LOG_CUDA "                    * rgb32f, bgr32f\n");
	LogError(LOG_CUDA "                    * rgba32f, bgra32f\n");

	return hipErrorInvalidValue;
}

hipError_t cudaMerge(void *input_color, void *input_alpha, void *output, size_t width, size_t height, imageFormat format)
{
	if( format == IMAGE_RGBA8 || format == IMAGE_BGRA8 )
		return launchMerge<uchar3, uchar, uchar4>((uchar3 *)input_color, (uchar *)input_alpha, (uchar4 *)output, width, height);
	else if( format == IMAGE_RGBA32F || format == IMAGE_BGRA32F )
		return launchMerge<float3, float, float4>((float3 *)input_color, (float *)input_alpha, (float4 *)output, width, height);

	LogError(LOG_CUDA "cudaMerge() -- invalid image format '%s'\n", imageFormatToStr(format));
	LogError(LOG_CUDA "                supported formats are:\n");
	LogError(LOG_CUDA "                    * rgba8, bgra8\n");
	LogError(LOG_CUDA "                    * rgba32f, bgra32f\n");

	return hipErrorInvalidValue;
}
