#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaFont.h"
#include "cudaVector.h"
#include "cudaOverlay.h"
#include "cudaMappedMemory.h"

#include "imageIO.h"
#include "filesystem.h"
#include "logging.h"

#define STBTT_STATIC
#define STB_TRUETYPE_IMPLEMENTATION

#include "../cpp/image/stb/stb_truetype.h"


//#define DEBUG_FONT


// Struct for one character to render
struct __align__(16) GlyphCommand
{
	short x;		// x coordinate origin in output image to begin drawing the glyph at 
	short y;		// y coordinate origin in output image to begin drawing the glyph at 
	short u;		// x texture coordinate in the baked font map where the glyph resides
	short v;		// y texture coordinate in the baked font map where the glyph resides 
	short width;	// width of the glyph in pixels
	short height;	// height of the glyph in pixels
};


// adaptFontSize
float adaptFontSize( uint32_t dimension )
{
	const float max_font = 32.0f;
	const float min_font = 28.0f;

	const uint32_t max_dim = 1536;
	const uint32_t min_dim = 768;

	if( dimension > max_dim )
		dimension = max_dim;

	if( dimension < min_dim )
		dimension = min_dim;

	const float dim_ratio = float(dimension - min_dim) / float(max_dim - min_dim);

	return min_font + dim_ratio * (max_font - min_font);
}


// constructor
cudaFont::cudaFont()
{
	mSize = 0.0f;
	
	mCommandCPU = NULL;
	mCommandGPU = NULL;
	mCmdIndex   = 0;

	mFontMapCPU = NULL;
	mFontMapGPU = NULL;

	mRectsCPU   = NULL;
	mRectsGPU   = NULL;
	mRectIndex  = 0;

	mFontMapWidth  = 256;
	mFontMapHeight = 256;
}



// destructor
cudaFont::~cudaFont()
{
	if( mRectsCPU != NULL )
	{
		CUDA(hipHostFree(mRectsCPU));
		
		mRectsCPU = NULL; 
		mRectsGPU = NULL;
	}

	if( mCommandCPU != NULL )
	{
		CUDA(hipHostFree(mCommandCPU));
		
		mCommandCPU = NULL; 
		mCommandGPU = NULL;
	}

	if( mFontMapCPU != NULL )
	{
		CUDA(hipHostFree(mFontMapCPU));
		
		mFontMapCPU = NULL; 
		mFontMapGPU = NULL;
	}
}


// Create
cudaFont* cudaFont::Create( float size )
{
	// default fonts	
	std::vector<std::string> fonts;
	
	fonts.push_back("/usr/share/fonts/truetype/dejavu/DejaVuSansMono.ttf");
	fonts.push_back("/usr/share/fonts/truetype/dejavu/DejaVuSans.ttf");

	return Create(fonts, size);
}


// Create
cudaFont* cudaFont::Create( const std::vector<std::string>& fonts, float size )
{
	const uint32_t numFonts = fonts.size();

	for( uint32_t n=0; n < numFonts; n++ )
	{
		cudaFont* font = Create(fonts[n].c_str(), size);

		if( font != NULL )
			return font;
	}

	return NULL;
}


// Create
cudaFont* cudaFont::Create( const char* font, float size )
{
	// verify parameters
	if( !font )
		return Create(size);

	// create new font
	cudaFont* c = new cudaFont();
	
	if( !c )
		return NULL;
		
	if( !c->init(font, size) )
	{
		delete c;
		return NULL;
	}

	return c;
}


// init
bool cudaFont::init( const char* filename, float size )
{
	// validate parameters
	if( !filename )
		return NULL;

	// verify that the font file exists and get its size
	const size_t ttf_size = fileSize(filename);

	if( !ttf_size )
	{
		LogError(LOG_CUDA "font doesn't exist or empty file '%s'\n", filename);
 		return false;
	}

	// allocate memory to store the font file
	void* ttf_buffer = malloc(ttf_size);

	if( !ttf_buffer )
	{
		LogError(LOG_CUDA "failed to allocate %zu byte buffer for reading '%s'\n", ttf_size, filename);
		return false;
	}

	// open the font file
	FILE* ttf_file = fopen(filename, "rb");

	if( !ttf_file )
	{
		LogError(LOG_CUDA "failed to open '%s' for reading\n", filename);
		free(ttf_buffer);
		return false;
	}

	// read the font file
	const size_t ttf_read = fread(ttf_buffer, 1, ttf_size, ttf_file);

	fclose(ttf_file);

	if( ttf_read != ttf_size )
	{
		LogError(LOG_CUDA "failed to read contents of '%s'\n", filename);
		LogError(LOG_CUDA "(read %zu bytes, expected %zu bytes)\n", ttf_read, ttf_size);

		free(ttf_buffer);
		return false;
	}

	// buffer that stores the coordinates of the baked glyphs
	stbtt_bakedchar bakeCoords[NumGlyphs];

	// increase the size of the bitmap until all the glyphs fit
	while(true)
	{
		// allocate memory for the packed font texture (alpha only)
		const size_t fontMapSize = mFontMapWidth * mFontMapHeight * sizeof(unsigned char);

		if( !cudaAllocMapped((void**)&mFontMapCPU, (void**)&mFontMapGPU, fontMapSize) )
		{
			LogError(LOG_CUDA "failed to allocate %zu bytes to store %ix%i font map\n", fontMapSize, mFontMapWidth, mFontMapHeight);
			free(ttf_buffer);
			return false;
		}

		// attempt to pack the bitmap
		const int result = stbtt_BakeFontBitmap((uint8_t*)ttf_buffer, 0, size, 
										mFontMapCPU, mFontMapWidth, mFontMapHeight,
									     FirstGlyph, NumGlyphs, bakeCoords);

		if( result == 0 )
		{
			LogError(LOG_CUDA "failed to bake font bitmap '%s'\n", filename);
			free(ttf_buffer);
			return false;
		}
		else if( result < 0 )
		{
			const int glyphsPacked = -result;

			if( glyphsPacked == NumGlyphs )
			{
				LogVerbose(LOG_CUDA "packed %u glyphs in %ux%u bitmap (font size=%.0fpx)\n", NumGlyphs, mFontMapWidth, mFontMapHeight, size);
				break;
			}

		#ifdef DEBUG_FONT
			LogDebug(LOG_CUDA "fit only %i of %u font glyphs in %ux%u bitmap\n", glyphsPacked, NumGlyphs, mFontMapWidth, mFontMapHeight);
		#endif

			CUDA(hipHostFree(mFontMapCPU));
		
			mFontMapCPU = NULL; 
			mFontMapGPU = NULL;

			mFontMapWidth *= 2;
			mFontMapHeight *= 2;

		#ifdef DEBUG_FONT
			LogDebug(LOG_CUDA "attempting to pack font with %ux%u bitmap...\n", mFontMapWidth, mFontMapHeight);
		#endif
			continue;
		}
		else
		{
		#ifdef DEBUG_FONT
			LogDebug(LOG_CUDA "packed %u glyphs in %ux%u bitmap (font size=%.0fpx)\n", NumGlyphs, mFontMapWidth, mFontMapHeight, size);
		#endif		
			break;
		}
	}

	// free the TTF font data
	free(ttf_buffer);

	// store texture baking coordinates
	for( uint32_t n=0; n < NumGlyphs; n++ )
	{
		mGlyphInfo[n].x = bakeCoords[n].x0;
		mGlyphInfo[n].y = bakeCoords[n].y0;

		mGlyphInfo[n].width  = bakeCoords[n].x1 - bakeCoords[n].x0;
		mGlyphInfo[n].height = bakeCoords[n].y1 - bakeCoords[n].y0;

		mGlyphInfo[n].xAdvance = bakeCoords[n].xadvance;
		mGlyphInfo[n].xOffset  = bakeCoords[n].xoff;
		mGlyphInfo[n].yOffset  = bakeCoords[n].yoff;

	#ifdef DEBUG_FONT
		// debug info
		const char c = n + FirstGlyph;
		LogDebug("Glyph %u: '%c' width=%hu height=%hu xOffset=%.0f yOffset=%.0f xAdvance=%0.1f\n", n, c, mGlyphInfo[n].width, mGlyphInfo[n].height, mGlyphInfo[n].xOffset, mGlyphInfo[n].yOffset, mGlyphInfo[n].xAdvance);
	#endif	
	}

	// allocate memory for GPU command buffer	
	if( !cudaAllocMapped(&mCommandCPU, &mCommandGPU, sizeof(GlyphCommand) * MaxCommands) )
		return false;
	
	// allocate memory for background rect buffers
	if( !cudaAllocMapped((void**)&mRectsCPU, (void**)&mRectsGPU, sizeof(float4) * MaxCommands) )
		return false;

	mSize = size;
	return true;
}


/*inline __host__ __device__ float4 operator*(float4 a, float4 b)
{
    return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}*/

inline __host__ __device__ float4 alpha_blend( const float4& bg, const float4& fg )
{
	const float alpha = fg.w / 255.0f;
	const float ialph = 1.0f - alpha;
	
	return make_float4(alpha * fg.x + ialph * bg.x,
				    alpha * fg.y + ialph * bg.y,
				    alpha * fg.z + ialph * bg.z,
				    bg.w);
} 


template<typename T>
__global__ void gpuOverlayText( unsigned char* font, int fontWidth, GlyphCommand* commands,
                                T* input, T* output, int imgWidth, int imgHeight, float4 color ) 
{
	const GlyphCommand cmd = commands[blockIdx.x];

	if( threadIdx.x >= cmd.width || threadIdx.y >= cmd.height )
		return;

	const int x = cmd.x + threadIdx.x;
	const int y = cmd.y + threadIdx.y;

	if( x < 0 || y < 0 || x >= imgWidth || y >= imgHeight )
		return;

	const int u = cmd.u + threadIdx.x;
	const int v = cmd.v + threadIdx.y;

	const float px_glyph = font[v * fontWidth + u];

	const float4 px_font = make_float4(px_glyph * color.x, px_glyph * color.y, px_glyph * color.z, px_glyph * color.w);
	const float4 px_in   = cast_vec<float4>(input[y * imgWidth + x]);

	output[y * imgWidth + x] = cast_vec<T>(alpha_blend(px_in, px_font));	 
}


// cudaOverlayText
hipError_t cudaOverlayText( unsigned char* font, const int2& maxGlyphSize, size_t fontMapWidth,
                             GlyphCommand* commands, size_t numCommands, const float4& fontColor, 
                             void* input, void* output, imageFormat format, size_t imgWidth, size_t imgHeight,
                             hipStream_t stream )	
{
	if( !font || !commands || !input || !output || numCommands == 0 || fontMapWidth == 0 || imgWidth == 0 || imgHeight == 0 )
		return hipErrorInvalidValue;

	const float4 color_scaled = make_float4( fontColor.x / 255.0f, fontColor.y / 255.0f, fontColor.z / 255.0f, fontColor.w / 255.0f );
	
	// setup arguments
	const dim3 block(maxGlyphSize.x, maxGlyphSize.y);
	const dim3 grid(numCommands);

	if( format == IMAGE_RGB8 )
		gpuOverlayText<uchar3><<<grid, block, 0, stream>>>(font, fontMapWidth, commands, (uchar3*)input, (uchar3*)output, imgWidth, imgHeight, color_scaled); 
	else if( format == IMAGE_RGBA8 )
		gpuOverlayText<uchar4><<<grid, block, 0, stream>>>(font, fontMapWidth, commands, (uchar4*)input, (uchar4*)output, imgWidth, imgHeight, color_scaled); 
	else if( format == IMAGE_RGB32F )
		gpuOverlayText<float3><<<grid, block, 0, stream>>>(font, fontMapWidth, commands, (float3*)input, (float3*)output, imgWidth, imgHeight, color_scaled); 
	else if( format == IMAGE_RGBA32F )
		gpuOverlayText<float4><<<grid, block, 0, stream>>>(font, fontMapWidth, commands, (float4*)input, (float4*)output, imgWidth, imgHeight, color_scaled); 
	else
		return hipErrorInvalidValue;

	return hipGetLastError();
}


// Overlay
bool cudaFont::OverlayText( void* image, imageFormat format, uint32_t width, uint32_t height, 
                            const std::vector< std::pair< std::string, int2 > >& strings, 
                            const float4& color, const float4& bg_color, int bg_padding,
                            hipStream_t stream )
{
	const uint32_t numStrings = strings.size();

	if( !image || width == 0 || height == 0 || numStrings == 0 )
		return false;

	if( format != IMAGE_RGB8 && format != IMAGE_RGBA8 && format != IMAGE_RGB32F && format != IMAGE_RGBA32F )
	{
		LogError(LOG_CUDA "cudaFont::OverlayText() -- unsupported image format (%s)\n", imageFormatToStr(format));
		LogError(LOG_CUDA "                           supported formats are:\n");
		LogError(LOG_CUDA "                              * rgb8\n");		
		LogError(LOG_CUDA "                              * rgba8\n");		
		LogError(LOG_CUDA "                              * rgb32f\n");		
		LogError(LOG_CUDA "                              * rgba32f\n");

		return false;
	}

	
	const bool has_bg = bg_color.w > 0.0f;
	int2 maxGlyphSize = make_int2(0,0);

	int numCommands = 0;
	int numRects = 0;
	int maxChars = 0;

	// find the bg rects and total char count
	for( uint32_t s=0; s < numStrings; s++ )
		maxChars += strings[s].first.size();

	// reset the buffer indices if we need the space
	if( mCmdIndex + maxChars >= MaxCommands )
		mCmdIndex = 0;

	if( has_bg && mRectIndex + numStrings >= MaxCommands )
		mRectIndex = 0;

	// generate glyph commands and bg rects
	for( uint32_t s=0; s < numStrings; s++ )
	{
		const uint32_t numChars = strings[s].first.size();
		
		if( numChars == 0 )
			continue;

		// determine the max 'height' of the string
		int maxHeight = 0;

		for( uint32_t n=0; n < numChars; n++ )
		{
			char c = strings[s].first[n];
			
			if( c < FirstGlyph || c > LastGlyph )
				continue;
			
			c -= FirstGlyph;

			const int yOffset = abs((int)mGlyphInfo[c].yOffset);

			if( maxHeight < yOffset )
				maxHeight = yOffset;
		}

	#ifdef DEBUG_FONT
		LogDebug(LOG_CUDA "max glyph height:  %i\n", maxHeight);
	#endif

		// get the starting position of the string
		int2 pos = strings[s].second;

		if( pos.x < 0 )
			pos.x = 0;

		if( pos.y < 0 )
			pos.y = 0;
		
		pos.y += maxHeight;

		// reset the background rect if needed
		if( has_bg )
			mRectsCPU[mRectIndex+numRects] = make_float4(width, height, 0, 0);

		// make a glyph command for each character
		for( uint32_t n=0; n < numChars; n++ )
		{
			char c = strings[s].first[n];
			
			// make sure the character is in range
			if( c < FirstGlyph || c > LastGlyph )
				continue;
			
			c -= FirstGlyph;	// rebase char against glyph 0
			
			// fill the next command
			GlyphCommand* cmd = ((GlyphCommand*)mCommandCPU) + mCmdIndex + numCommands;

			cmd->x = pos.x;
			cmd->y = pos.y + mGlyphInfo[c].yOffset;
			cmd->u = mGlyphInfo[c].x;
			cmd->v = mGlyphInfo[c].y;

			cmd->width  = mGlyphInfo[c].width;
			cmd->height = mGlyphInfo[c].height;
		
			// advance the text position
			pos.x += mGlyphInfo[c].xAdvance;

			// track the maximum glyph size
			if( maxGlyphSize.x < mGlyphInfo[c].width )
				maxGlyphSize.x = mGlyphInfo[c].width;

			if( maxGlyphSize.y < mGlyphInfo[c].height )
				maxGlyphSize.y = mGlyphInfo[c].height;

			// expand the background rect
			if( has_bg )
			{
				float4* rect = mRectsCPU + mRectIndex + numRects;

				if( cmd->x < rect->x )
					rect->x = cmd->x;

				if( cmd->y < rect->y )
					rect->y = cmd->y;

				const float x2 = cmd->x + cmd->width;
				const float y2 = cmd->y + cmd->height;

				if( x2 > rect->z )
					rect->z = x2;

				if( y2 > rect->w )
					rect->w = y2;
			}

			numCommands++;
		}

		if( has_bg )
		{
			float4* rect = mRectsCPU + mRectIndex + numRects;

			// apply padding
			rect->x -= bg_padding;
			rect->y -= bg_padding;
			rect->z += bg_padding;
			rect->w += bg_padding;

			numRects++;
		}
	}

#ifdef DEBUG_FONT
	LogDebug(LOG_CUDA "max glyph size is %ix%i\n", maxGlyphSize.x, maxGlyphSize.y);
#endif

	// draw background rects
	if( has_bg && numRects > 0 )
		CUDA(cudaRectFill(image, image, width, height, format, mRectsGPU + mRectIndex, numRects, bg_color, stream));

	// draw text characters
	CUDA(cudaOverlayText(mFontMapGPU, maxGlyphSize, mFontMapWidth,
                         ((GlyphCommand*)mCommandGPU) + mCmdIndex, numCommands, 
                         color, image, image, format, width, height, stream));
			
	// advance the buffer indices
	mCmdIndex += numCommands;
	mRectIndex += numRects;
		   
	return true;
}


// Overlay
bool cudaFont::OverlayText( void* image, imageFormat format, uint32_t width, uint32_t height, 
                            const char* str, int x, int y, const float4& color, const float4& bg_color, 
                            int bg_padding, hipStream_t stream )
{
	if( !str )
		return NULL;
		
	std::vector< std::pair< std::string, int2 > > list;
	
	list.push_back( std::pair< std::string, int2 >( str, make_int2(x,y) ));

	return OverlayText(image, format, width, height, list, color, bg_color, bg_padding, stream);
}


// TextExtents
int4 cudaFont::TextExtents( const char* str, int x, int y )
{
	if( !str )
		return make_int4(0,0,0,0);

	const size_t numChars = strlen(str);

	// determine the max 'height' of the string
	int maxHeight = 0;

	for( uint32_t n=0; n < numChars; n++ )
	{
		char c = str[n];
		
		if( c < FirstGlyph || c > LastGlyph )
			continue;
		
		c -= FirstGlyph;

		const int yOffset = abs((int)mGlyphInfo[c].yOffset);

		if( maxHeight < yOffset )
			maxHeight = yOffset;
	}

	// get the starting position of the string
	int2 pos = make_int2(x,y);

	if( pos.x < 0 )
		pos.x = 0;

	if( pos.y < 0 )
		pos.y = 0;
	
	pos.y += maxHeight;


	// find the extents of the string
	for( uint32_t n=0; n < numChars; n++ )
	{
		char c = str[n];
		
		// make sure the character is in range
		if( c < FirstGlyph || c > LastGlyph )
			continue;
		
		c -= FirstGlyph;	// rebase char against glyph 0
		
		// advance the text position
		pos.x += mGlyphInfo[c].xAdvance;
	}

	return make_int4(x, y, pos.x, pos.y);
}
	


				
	
