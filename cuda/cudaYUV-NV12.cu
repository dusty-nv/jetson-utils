#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaYUV.h"


#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff

#define MUL(x,y)    (x*y)



__constant__ uint32_t constAlpha;
__constant__ float  constHueColorSpaceMat[9];



__device__ void YUV2RGB(uint32_t *yuvi, float *red, float *green, float *blue)
{
   

    // Prepare for hue adjustment
    /*
	 float luma, chromaCb, chromaCr;

	luma     = (float)yuvi[0];
    chromaCb = (float)((int)yuvi[1] - 512.0f);
    chromaCr = (float)((int)yuvi[2] - 512.0f);

    // Convert YUV To RGB with hue adjustment
    *red  = MUL(luma,     constHueColorSpaceMat[0]) +
            MUL(chromaCb, constHueColorSpaceMat[1]) +
            MUL(chromaCr, constHueColorSpaceMat[2]);
    *green= MUL(luma,     constHueColorSpaceMat[3]) +
            MUL(chromaCb, constHueColorSpaceMat[4]) +
            MUL(chromaCr, constHueColorSpaceMat[5]);
    *blue = MUL(luma,     constHueColorSpaceMat[6]) +
            MUL(chromaCb, constHueColorSpaceMat[7]) +
            MUL(chromaCr, constHueColorSpaceMat[8]);*/

	const float luma = float(yuvi[0]);
	const float u    = float(yuvi[1]) - 512.0f;
	const float v    = float(yuvi[2]) - 512.0f;

   /*R = Y + 1.140V
   G = Y - 0.395U - 0.581V
   B = Y + 2.032U*/

	/**green = luma + 1.140f * v;
	*blue  = luma - 0.395f * u - 0.581f * v;
	*red   = luma + 2.032f * u;*/

	*red    = luma + 1.140f * v;
	*green  = luma - 0.395f * u - 0.581f * v;
	*blue   = luma + 2.032f * u;
}


__device__ uint32_t RGBAPACK_8bit(float red, float green, float blue, uint32_t alpha)
{
    uint32_t ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 255.0f);
    green = min(max(green, 0.0f), 255.0f);
    blue  = min(max(blue,  0.0f), 255.0f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = ((((uint32_t)red)   << 24) |
                 (((uint32_t)green) << 16) |
		       (((uint32_t)blue)  <<  8) | (uint32_t)alpha);

    return  ARGBpixel;
}


__device__ uint32_t RGBAPACK_10bit(float red, float green, float blue, uint32_t alpha)
{
    uint32_t ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 1023.f);
    green = min(max(green, 0.0f), 1023.f);
    blue  = min(max(blue,  0.0f), 1023.f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = ((((uint32_t)red   >> 2) << 24) |
                 (((uint32_t)green >> 2) << 16) |
                 (((uint32_t)blue  >> 2) <<  8) | (uint32_t)alpha);

    return  ARGBpixel;
}


// CUDA kernel for outputing the final ARGB output from NV12;
/*extern "C"*/
__global__ void Passthru(uint32_t *srcImage,   size_t nSourcePitch,
                         uint32_t *dstImage,   size_t nDestPitch,
                         uint32_t width,       uint32_t height)
{
    int x, y;
    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = ((width) + 63) & ~63;
    uint32_t dstImagePitch   = nDestPitch >> 2;
    uint8_t *srcImageU8     = (uint8_t *)srcImage;

    processingPitch = nSourcePitch;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]);
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]);

    // this steps performs the color conversion
    float luma[2];

    luma[0]   = (yuv101010Pel[0]        & 0x00FF);
    luma[1]   = (yuv101010Pel[1]        & 0x00FF);

    // Clamp the results to RGBA
    dstImage[y * dstImagePitch + x     ] = RGBAPACK_8bit(luma[0], luma[0], luma[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_8bit(luma[1], luma[1], luma[1], constAlpha);
}


// CUDA kernel for outputing the final ARGB output from NV12;
/*extern "C"*/
__global__ void NV12ToARGB(uint32_t *srcImage,     size_t nSourcePitch,
                           uint32_t *dstImage,     size_t nDestPitch,
                           uint32_t width,         uint32_t height)
{
    int x, y;
    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = ((width) + 63) & ~63;
    uint32_t dstImagePitch   = nDestPitch >> 2;
    uint8_t *srcImageU8     = (uint8_t *)srcImage;

    processingPitch = nSourcePitch;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset    = processingPitch * height;
    int y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        uint32_t chromaCb;
        uint32_t chromaCr;

        chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    // this steps performs the color conversion
    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] &   COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] &   COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA
    dstImage[y * dstImagePitch + x     ] = RGBAPACK_10bit(red[0], green[0], blue[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1 ] = RGBAPACK_10bit(red[1], green[1], blue[1], constAlpha);
}


bool nv12ColorspaceSetup = false;


// cudaNV12ToARGB32
hipError_t cudaNV12ToRGBA( uint8_t* srcDev, size_t srcPitch, uchar4* destDev, size_t destPitch, size_t width, size_t height )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	if( srcPitch == 0 || destPitch == 0 || width == 0 || height == 0 )
		return hipErrorInvalidValue;

	if( !nv12ColorspaceSetup )
		cudaNV12SetupColorspace();

	const dim3 blockDim(32,16,1);
	const dim3 gridDim((width+(2*blockDim.x-1))/(2*blockDim.x), (height+(blockDim.y-1))/blockDim.y, 1);

	NV12ToARGB<<<gridDim, blockDim>>>( (uint32_t*)srcDev, srcPitch, (uint32_t*)destDev, destPitch, width, height );
	
	return CUDA(hipGetLastError());
}

hipError_t cudaNV12ToRGBA( uint8_t* srcDev, uchar4* destDev, size_t width, size_t height )
{
	return cudaNV12ToRGBA(srcDev, width * sizeof(uint8_t), destDev, width * sizeof(uchar4), width, height);
}


//-------------------------------------------------------------------------------------------------------------------------

__global__ void NV12ToRGBAf(uint32_t* srcImage,  size_t nSourcePitch,
                           float4* dstImage,     size_t nDestPitch,
                           uint32_t width,       uint32_t height)
{
    int x, y;
    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = ((width) + 63) & ~63;
    uint8_t *srcImageU8     = (uint8_t *)srcImage;

    processingPitch = nSourcePitch;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

#if 1	
    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset    = processingPitch * height;
    int y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        uint32_t chromaCb;
        uint32_t chromaCr;

        chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    // this steps performs the color conversion
    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] &   COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] &   COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA
	//printf("cuda thread %i %i  %f %f %f\n", x, y, red[0], green[0], blue[0]);

	const float s = 1.0f / 1024.0f * 255.0f;

	dstImage[y * width + x]     = make_float4(red[0] * s, green[0] * s, blue[0] * s, 1.0f);
	dstImage[y * width + x + 1] = make_float4(red[1] * s, green[1] * s, blue[1] * s, 1.0f);
#else
	//printf("cuda thread %i %i  %i %i \n", x, y, width, height);
		
	dstImage[y * width + x]     = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
	dstImage[y * width + x + 1] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
#endif
}


__global__ void NV12ToBGR8(uint32_t* srcImage,  size_t nSourcePitch,
                           uchar3* dstImage,     size_t nDestPitch,
                           uint32_t width,       uint32_t height)
{
    int x, y;
    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = ((width) + 63) & ~63;
    uint8_t *srcImageU8     = (uint8_t *)srcImage;

    processingPitch = nSourcePitch;

    // Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y *  blockDim.y       +  threadIdx.y;

    if (x >= width)
        return; //x = width - 1;

    if (y >= height)
        return; // y = height - 1;

#if 1
    // Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
    // if we move to texture we could read 4 luminance values
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset    = processingPitch * height;
    int y_chroma = y >> 1;

    if (y & 1)  // odd scanline ?
    {
        uint32_t chromaCb;
        uint32_t chromaCr;

        chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
        chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    // this steps performs the color conversion
    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] &   COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] &   COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA
        //printf("cuda thread %i %i  %f %f %f\n", x, y, red[0], green[0], blue[0]);

        const float s = 1.0f / 1024.0f * 255.0f;

        dstImage[y * width + x]     = make_uchar3(blue[0] * s, green[0] * s, red[0] * s);
        dstImage[y * width + x + 1] = make_uchar3(blue[1] * s, green[1] * s, red[1] * s);
#else
        //printf("cuda thread %i %i  %i %i \n", x, y, width, height);
                
        dstImage[y * width + x]     = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
        dstImage[y * width + x + 1] = make_float4(1.0f, 0.0f, 0.0f, 1.0f);
#endif
}



// cudaNV12ToRGBA
hipError_t cudaNV12ToRGBA32( uint8_t* srcDev, size_t srcPitch, float4* destDev, size_t destPitch, size_t width, size_t height )
{
	if( !srcDev || !destDev )
		return hipErrorInvalidDevicePointer;

	if( srcPitch == 0 || destPitch == 0 || width == 0 || height == 0 )
		return hipErrorInvalidValue;

	if( !nv12ColorspaceSetup )
		cudaNV12SetupColorspace();

	const dim3 blockDim(8,8,1);
	//const dim3 gridDim((width+(2*blockDim.x-1))/(2*blockDim.x), (height+(blockDim.y-1))/blockDim.y, 1);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height, blockDim.y), 1);

	NV12ToRGBAf<<<gridDim, blockDim>>>( (uint32_t*)srcDev, srcPitch, destDev, destPitch, width, height );
	
	return CUDA(hipGetLastError());
}

hipError_t cudaNV12ToRGBA32( uint8_t* srcDev, float4* destDev, size_t width, size_t height )
{
	return cudaNV12ToRGBA32(srcDev, width * sizeof(uint8_t), destDev, width * sizeof(float4), width, height);
}



hipError_t cudaNV12ToBGR8( uint8_t* srcDev, size_t srcPitch, uchar3* destDev, size_t destPitch, size_t width, size_t height )
{
        if( !srcDev || !destDev )
                return hipErrorInvalidDevicePointer;

        if( srcPitch == 0 || destPitch == 0 || width == 0 || height == 0 )
                return hipErrorInvalidValue;

        if( !nv12ColorspaceSetup )
                cudaNV12SetupColorspace();

        const dim3 blockDim(8,8,1);
        //const dim3 gridDim((width+(2*blockDim.x-1))/(2*blockDim.x), (height+(blockDim.y-1))/blockDim.y, 1);
        const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height, blockDim.y), 1);

        NV12ToBGR8<<<gridDim, blockDim>>>( (uint32_t*)srcDev, srcPitch, destDev, destPitch, width, height );

        return CUDA(hipGetLastError());
}

hipError_t cudaNV12ToBGR8( uint8_t* srcDev, uchar3* destDev, size_t width, size_t height )
{
        return cudaNV12ToBGR8(srcDev, width * sizeof(uint8_t), destDev, width * sizeof(float4), width, height);
}



// cudaNV12SetupColorspace
hipError_t cudaNV12SetupColorspace( float hue )
{
	const float hueSin = sin(hue);
	const float hueCos = cos(hue);

	float hueCSC[9];

	const bool itu601 = false;

	if( itu601 /*CSC == ITU601*/)
	{
		//CCIR 601
		hueCSC[0] = 1.1644f;
		hueCSC[1] = hueSin * 1.5960f;
		hueCSC[2] = hueCos * 1.5960f;
		hueCSC[3] = 1.1644f;
		hueCSC[4] = (hueCos * -0.3918f) - (hueSin * 0.8130f);
		hueCSC[5] = (hueSin *  0.3918f) - (hueCos * 0.8130f);
		hueCSC[6] = 1.1644f;
		hueCSC[7] = hueCos *  2.0172f;
		hueCSC[8] = hueSin * -2.0172f;
	}
	else /*if(CSC == ITU709)*/
	{
		//CCIR 709
		hueCSC[0] = 1.0f;
		hueCSC[1] = hueSin * 1.57480f;
		hueCSC[2] = hueCos * 1.57480f;
		hueCSC[3] = 1.0;
		hueCSC[4] = (hueCos * -0.18732f) - (hueSin * 0.46812f);
		hueCSC[5] = (hueSin *  0.18732f) - (hueCos * 0.46812f);
		hueCSC[6] = 1.0f;
		hueCSC[7] = hueCos *  1.85560f;
		hueCSC[8] = hueSin * -1.85560f;
	}


	if( CUDA_FAILED(hipMemcpyToSymbol(HIP_SYMBOL(constHueColorSpaceMat), hueCSC, sizeof(float) * 9)) )
		return hipErrorInvalidSymbol;

	uint32_t cudaAlpha = ((uint32_t)0xff<< 24);

	if( CUDA_FAILED(hipMemcpyToSymbol(HIP_SYMBOL(constAlpha), &cudaAlpha, sizeof(uint32_t))) )
		return hipErrorInvalidSymbol;

	nv12ColorspaceSetup = true;
	return hipSuccess;
}

